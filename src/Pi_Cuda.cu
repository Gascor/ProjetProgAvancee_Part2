#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <corecrt_math_defines.h>

// Implémentation manuelle de atomicAdd pour les doubles
__device__ void atomicAddDouble(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
}

// Kernel pour initialiser les états RNG
__global__ void init_rng(hiprandState *states, unsigned long long seed, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// Kernel pour calculer Pi
__global__ void compute_pi(double *estimate, hiprandState *states, unsigned long long n_samples) {
    extern __shared__ unsigned long long local_counts[]; // Mémoire partagée
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    unsigned long long count = 0;
    double x, y, z;

    // Récupérez l'état RNG pré-initialisé
    hiprandState local_state = states[idx];

    // Calcul des points dans le cercle
    for (unsigned long long i = idx; i < n_samples; i += stride) {
        x = hiprand_uniform_double(&local_state);
        y = hiprand_uniform_double(&local_state);
        z = x * x + y * y;
        if (z <= 1.0) count++;
    }

    states[idx] = local_state;  // Sauvegardez l'état
    local_counts[threadIdx.x] = count;
    __syncthreads();

    // Réduction locale dans le bloc
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            local_counts[threadIdx.x] += local_counts[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Ajoutez le total du bloc au résultat global
    if (threadIdx.x == 0) {
        atomicAddDouble(estimate, (double)local_counts[0]);
    }
}

// Fonction principale
int main(int argc, char *argv[]) {
    FILE *fp = fopen("pi_results_optimized3suite.csv", "w");
    fprintf(fp, "PI;Difference;Error;Ntot;AvailableProcessors;TimeDuration(ms);\n");

    double *d_estimate, h_estimate;
    hiprandState *d_states;

    int threadsPerBlock = 1024;
    int numBlocks = 8096;

    hipMalloc((void **)&d_estimate, sizeof(double));
    hipMalloc((void **)&d_states, numBlocks * threadsPerBlock * sizeof(hiprandState));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Initialisation des RNG
    init_rng<<<numBlocks, threadsPerBlock>>>(d_states, time(NULL), numBlocks * threadsPerBlock);
    hipDeviceSynchronize();

    for (int power = 5; power <= 44; ++power) {
        unsigned long long n_samples = (1ULL << power);

        for (int test = 0; test < 5; ++test) {
            hipMemset(d_estimate, 0, sizeof(double));

            hipEventRecord(start);
            compute_pi<<<numBlocks, threadsPerBlock, threadsPerBlock * sizeof(unsigned long long)>>>(d_estimate, d_states, n_samples);
            hipEventRecord(stop);

            hipDeviceSynchronize();

            hipMemcpy(&h_estimate, d_estimate, sizeof(double), hipMemcpyDeviceToHost);
            printf("Estimate from GPU: %f\n", h_estimate); // Affiche le résultat brut

            h_estimate = (h_estimate / n_samples) * 4.0;
            printf("Calculated PI: %f\n", h_estimate);

            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);

            double pi_error = fabs(h_estimate - M_PI);
            fprintf(fp, "%.10f;%.10f;%.10f;%llu;1;%.3f;\n", h_estimate, h_estimate - M_PI, pi_error / M_PI, n_samples, milliseconds);
        }
    }

    hipFree(d_estimate);
    hipFree(d_states);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    fclose(fp);

    return 0;
}
